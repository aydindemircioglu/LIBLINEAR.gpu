
#include "gpuHelpers.h"
#include <iostream>
#include <hip/hip_runtime_api.h>



void checkCublas(hipblasStatus_t  s) 
{
	if (s != HIPBLAS_STATUS_SUCCESS) {
		switch (s) {
			case HIPBLAS_STATUS_ALLOC_FAILED:
				std::cerr  << "HIPBLAS_STATUS_ALLOC_FAILED" ;
				break;
			case HIPBLAS_STATUS_ARCH_MISMATCH:
				std::cerr  << "HIPBLAS_STATUS_ARCH_MISMATCH" ;
				break;
			case HIPBLAS_STATUS_EXECUTION_FAILED:
				std::cerr  << "HIPBLAS_STATUS_EXECUTION_FAILED" ;
				break;
			case HIPBLAS_STATUS_INTERNAL_ERROR:
				std::cerr  << "HIPBLAS_STATUS_INTERNAL_ERROR" ;
				break;
			case HIPBLAS_STATUS_INVALID_VALUE:
				std::cerr  << "HIPBLAS_STATUS_INVALID_VALUE" ;
				break;
			case HIPBLAS_STATUS_MAPPING_ERROR:
				std::cerr  << "HIPBLAS_STATUS_MAPPING_ERROR" ;
				break;
			case HIPBLAS_STATUS_NOT_INITIALIZED:
				std::cerr  << "HIPBLAS_STATUS_NOT_INITIALIZED" ;
				break;
			default:
				std::cerr  << "CUBLAS_UNKNOWN_ERROR" ;
		}
	};
}




void checkCusparse(hipsparseStatus_t s) {
    if (s != HIPSPARSE_STATUS_SUCCESS) {
        switch (s) {
            case HIPSPARSE_STATUS_NOT_INITIALIZED:
                std::cerr  << "HIPSPARSE_STATUS_NOT_INITIALIZED" ;
                break;
            case HIPSPARSE_STATUS_ALLOC_FAILED:
                std::cerr  << "HIPSPARSE_STATUS_ALLOC_FAILED" ;
                break;
            case HIPSPARSE_STATUS_INVALID_VALUE:
                std::cerr  << "HIPSPARSE_STATUS_INVALID_VALUE" ;
                break;
            case HIPSPARSE_STATUS_ARCH_MISMATCH:
                std::cerr  << "HIPSPARSE_STATUS_ARCH_MISMATCH" ;
                break;
            case HIPSPARSE_STATUS_MAPPING_ERROR:
                std::cerr  << "HIPSPARSE_STATUS_MAPPING_ERROR" ;
                break;
            case HIPSPARSE_STATUS_EXECUTION_FAILED:
                std::cerr  << "HIPSPARSE_STATUS_EXECUTION_FAILED" ;
                break;
            case HIPSPARSE_STATUS_INTERNAL_ERROR:
                std::cerr  << "HIPSPARSE_STATUS_INTERNAL_ERROR" ;
                break;
            case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
                std::cerr  << "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED" ;
                break;
            default:
                std::cerr  << "unknown CUSPARSE error" ;
        }
    }
 };
	
	
	
unsigned long long getGFlopsOfDeviceId (unsigned int deviceID) 
{
    int sm_per_multiproc  = 0;
    unsigned long long gflops = 0;
    hipDeviceProp_t deviceProp;
    
	hipGetDeviceProperties(&deviceProp, deviceID);

	// If this GPU is not running on Compute Mode prohibited, then we can add it to the list
	if (deviceProp.computeMode != hipComputeModeProhibited) {
		if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
			sm_per_multiproc = 1;
		} else {
			sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
		}
		gflops = (unsigned long long) deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
	}

    return gflops;
}




// This function returns the best GPU (with maximum GFLOPS)
int getMaxGflopsDeviceId()
{
    int current_device     = 0;
    int max_perf_device    = 0;
    int device_count       = 0;
    int devices_prohibited = 0;
    
    unsigned long long max_compute_perf = 0;
    hipGetDeviceCount(&device_count);
    
    checkCudaErrors(hipGetDeviceCount(&device_count));

    if (device_count == 0) {
        fprintf(stderr, "gpuGetMaxGflopsDeviceId() CUDA error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }

    if (devices_prohibited == device_count) {
    	fprintf(stderr, "gpuGetMaxGflopsDeviceId() CUDA error: all devices have compute mode prohibited.\n");
    	exit(EXIT_FAILURE);
    }

    // Find the best CUDA capable GPU device
    current_device = 0;

    while (current_device < device_count) {
		unsigned long long compute_perf = getGFlopsOfDeviceId (current_device);
		if (compute_perf  > max_compute_perf) {
			max_compute_perf  = compute_perf;
			max_perf_device   = current_device;
		}
        ++current_device;
    }

    return max_perf_device;
}

